#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include "./headers/kernels.h"
#include "./headers/kernels_proxy.h"

void sum_along_channels(float *act, float *batches, int nv, int Kin, int batchsize)
{
    float *d_act, *d_batches;

    size_t size_act = Kin;
    size_t size_batches = nv * nv * Kin * batchsize;

    hipMalloc((void**)&d_act, size_act * sizeof(float));
    hipMalloc((void**)&d_batches, size_batches * sizeof(float));

    hipMemcpy(d_act, act, size_act * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_batches, batches, size_batches * sizeof(float), hipMemcpyHostToDevice);

    init_with_zero_kernel<<< size_act / TPB_1d + 1, TPB_1d >>>(d_act, size_act);
    
    dim3 blockSize( TPB_2d, TPB_2d);
    dim3 gridSize( Kin, batchsize);
    sum_along_channels_kernel<<< gridSize, blockSize >>>(d_act, d_batches, nv, Kin, batchsize);
    
    hipMemcpy(act, d_act, size_act * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceReset();
}


void filter3d_valid(float *H, float *V, float *W, int nv, int nw, int Kin, int Kout){
	float *d_V, *d_H, *d_W;

	const int nh = nv - nw + 1;

	hipMalloc((void**)&d_H, nh * nh * Kout * sizeof(float));
	hipMalloc((void**)&d_V, nv * nv * Kin  * sizeof(float));
	hipMalloc((void**)&d_W, nw * nw * Kin * Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, nv * nv * Kin  * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, nw * nw * Kin  * Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout);	

	filter3d_valid_kernel<<< gridSize, blockSize >>>(d_H, d_V, d_W, nv, nw);
	
	hipMemcpy(H, d_H, nh * nh * Kout * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void filter4d_valid(float *H, float *V, float *W, int nv, int nw, int Kin, int Kout, int batchsize){
	float *d_V, *d_H, *d_W;

	const int nh = nv - nw + 1;

	hipMalloc((void**)&d_H, nh * nh * Kout * batchsize * sizeof(float));
	hipMalloc((void**)&d_V, nv * nv * Kin  * batchsize * sizeof(float));
	hipMalloc((void**)&d_W, nw * nw * Kin * Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, nv * nv * Kin  * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, nw * nw * Kin * Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout, batchsize);	

	filter4d_valid_kernel<<< gridSize, blockSize >>>(d_H, d_V, d_W, nv, nw);
	
	hipMemcpy(H, d_H, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void filter3d_full(float *V, float *H, float *W, int nh, int nw, int Kin, int Kout){
	float *d_V, *d_H, *d_W;

	const int nv = nh + nw - 1;

	hipMalloc((void**)&d_H, nh * nh * Kout * sizeof(float));
	hipMalloc((void**)&d_V, nv * nv * Kin  * sizeof(float));
	hipMalloc((void**)&d_W, nw * nw * Kin * Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, nv * nv * Kin  * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, nw * nw * Kin  * Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout);	

	filter3d_full_kernel<<< gridSize, blockSize >>>(d_V, d_H, d_W, nh, nw);
	
	hipMemcpy(V, d_V, nv * nv * Kin * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void filter4d_full(float *V, float *H, float *W, int nh, int nw, int Kin, int Kout, int batchsize){
	float *d_V, *d_H, *d_W;

	const int nv = nh + nw - 1;

	hipMalloc((void**)&d_H, nh * nh * Kout * batchsize * sizeof(float));
	hipMalloc((void**)&d_V, nv * nv * Kin * batchsize  * sizeof(float));
	hipMalloc((void**)&d_W, nw * nw * Kin * Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, nv * nv * Kin  * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, nw * nw * Kin  * Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout, batchsize);	

	filter4d_full_kernel<<< gridSize, blockSize >>>(d_V, d_H, d_W, nh, nw);
	
	hipMemcpy(V, d_V, nv * nv * Kin * batchsize * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void conv3d_valid(float *H, float *V, float *W, int nv, int nw, int Kin, int Kout){
	float *d_V, *d_H, *d_W;

	const int nh = nv - nw + 1;

	hipMalloc((void**)&d_H, nh * nh * Kout * sizeof(float));
	hipMalloc((void**)&d_V, nv * nv * Kin  * sizeof(float));
	hipMalloc((void**)&d_W, nw * nw * Kin * Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, nv * nv * Kin  * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, nw * nw * Kin  * Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout);	

	conv3d_valid_kernel<<< gridSize, blockSize >>>(d_H, d_V, d_W, nv, nw);
	
	hipMemcpy(H, d_H, nh * nh * Kout * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void conv4d_valid(float *H, float *V, float *W, int nv, int nw, int Kin, int Kout, int batchsize){
	float *d_V, *d_H, *d_W;

	const int nh = nv - nw + 1;

	hipMalloc((void**)&d_H, nh * nh * Kout * batchsize * sizeof(float));
	hipMalloc((void**)&d_V, nv * nv * Kin  * batchsize * sizeof(float));
	hipMalloc((void**)&d_W, nw * nw * Kin * Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, nv * nv * Kin  * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, nw * nw * Kin * Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout, batchsize);	

	conv4d_valid_kernel<<< gridSize, blockSize >>>(d_H, d_V, d_W, nv, nw);
	
	hipMemcpy(H, d_H, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void conv3d_full(float *V, float *H, float *W, int nh, int nw, int Kin, int Kout){
	float *d_V, *d_H, *d_W;

	const int nv = nh + nw - 1;

	hipMalloc((void**)&d_H, nh * nh * Kout * sizeof(float));
	hipMalloc((void**)&d_V, nv * nv * Kin  * sizeof(float));
	hipMalloc((void**)&d_W, nw * nw * Kin * Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, nv * nv * Kin  * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, nw * nw * Kin  * Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout);	

	conv3d_full_kernel<<< gridSize, blockSize >>>(d_V, d_H, d_W, nh, nw);
	
	hipMemcpy(V, d_V, nv * nv * Kin * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void conv4d_full(float *V, float *H, float *W, int nh, int nw, int Kin, int Kout, int batchsize){
	float *d_V, *d_H, *d_W;

	const int nv = nh + nw - 1;

	hipMalloc((void**)&d_H, nh * nh * Kout * batchsize * sizeof(float));
	hipMalloc((void**)&d_V, nv * nv * Kin * batchsize  * sizeof(float));
	hipMalloc((void**)&d_W, nw * nw * Kin * Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, nv * nv * Kin  * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, nw * nw * Kin  * Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout, batchsize);	

	conv4d_full_kernel<<< gridSize, blockSize >>>(d_V, d_H, d_W, nh, nw);
	
	hipMemcpy(V, d_V, nv * nv * Kin * batchsize * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void mapping(float *H, float *hb, int nh, int Kout, int batchsize) {
	float *d_H, *d_hb;

	hipMalloc((void**)&d_H, nh * nh * Kout * batchsize * sizeof(float));
	hipMalloc((void**)&d_hb, Kout * sizeof(float));

	hipMemcpy(d_H, H, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_hb, hb, Kout * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kout, batchsize);

	mapping_kernel<<< gridSize, blockSize >>>(d_H, d_hb, nh);

	hipMemcpy(H, d_H, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void pooling(float *P, float *HS, int C, int nh, int Kout, int batchsize){
	float *d_P, *d_HS;

	const int np = nh / C;

	hipMalloc((void**)&d_P,  np * np * Kout * batchsize * sizeof(float));
	hipMalloc((void**)&d_HS, nh * nh * Kout * batchsize * sizeof(float));

	hipMemcpy(d_P, P,   np * np * Kout * batchsize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_HS, HS, nh * nh * Kout * batchsize * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kout, batchsize);

	pooling_kernel<<< gridSize, blockSize >>>(d_P, d_HS, C, nh);

	hipMemcpy(P, d_P, np * np * Kout * batchsize * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

void prod3d(float *prod, float *V, float *H, int nv, int nh, int Kin, int Kout, int batchsize) {
	float *d_prod, *d_V, *d_H;

	const int nw = nv - nh + 1;

	int size_of_prod = nw * nw * Kin * Kout;
	int size_of_V = nv * nv * Kin * batchsize;
	int size_of_H = nh * nh * Kout * batchsize;

	hipMalloc((void**)&d_prod, size_of_prod * sizeof(float));
	hipMalloc((void**)&d_V,  size_of_V * sizeof(float));
	hipMalloc((void**)&d_H,  size_of_H * sizeof(float));

	hipMemcpy(d_prod, prod,   size_of_prod * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_V, V, size_of_V * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_H, H, size_of_H * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(TPB_2d, TPB_2d);
	dim3 gridSize(Kin, Kout, batchsize);

	prod3d_kernel<<< gridSize, blockSize >>>(d_prod, d_V, d_H, nv, nh);

	hipMemcpy(prod, d_prod, size_of_prod * sizeof(float), hipMemcpyDeviceToHost);

	hipDeviceReset();
}

