#include <stdlib.h>
#include "./headers/utils.h"
#include "./headers/crbm.h"

void crbmTrain_proxy(float *W, float *vb, float *hb, float *patches, Parameters pars)
{
    float *d_W, *d_vb, *d_hb, *d_patches;

    size_t num_saves = pars.maxIter/pars.SAVE_PER_ITERS + 1;
    size_t size_W  = pars.nw * pars.nw * pars.Kin * pars.Kout;
    size_t size_vb = pars.Kin;
    size_t size_hb = pars.Kout;
    size_t size_patches = pars.nv * pars.nv * pars.Kin * pars.numsamples;

    //init_gaussian(W, 0, 0.1, size_W);
    //init_const(vb,  0.0, size_vb); 
    //init_const(hb, -0.1, size_hb);

    hipMalloc((void**)&d_W,  size_W  * num_saves * sizeof(float));
    hipMalloc((void**)&d_vb, size_vb * num_saves * sizeof(float));
    hipMalloc((void**)&d_hb, size_hb * num_saves * sizeof(float));
    hipMalloc((void**)&d_patches, size_patches * sizeof(float));

    hipMemcpy(d_W,  W,  size_W  * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_hb, hb, size_hb * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vb, vb, size_vb * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_patches, patches, size_patches * sizeof(float), hipMemcpyHostToDevice);

    crbmTrain(d_W, d_vb, d_hb, d_patches, pars);

    hipMemcpy(W,  d_W,  size_W  * num_saves * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vb, d_vb, size_vb * num_saves * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(hb, d_hb, size_hb * num_saves * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_W);
    hipFree(d_vb);
    hipFree(d_hb);
    hipFree(d_patches);
}

void crbmInfer_proxy(float *H, float *HS, float *V, float *W, float *hb, Parameters pars)
{
    float *d_H, *d_HS, *d_V, *d_W, *d_hb;
	
	size_t size_V_batch = pars.nv * pars.nv * pars.Kin  * pars.batchsize;
	size_t size_H_batch = pars.nh * pars.nh * pars.Kout * pars.batchsize;
    size_t size_W  = pars.nw * pars.nw * pars.Kin * pars.Kout;
    size_t size_hb = pars.Kout;

	hipMalloc((void**)&d_H,  size_H_batch  * sizeof(float));
	hipMalloc((void**)&d_HS, size_H_batch  * sizeof(float));
	hipMalloc((void**)&d_V,  size_V_batch  * sizeof(float));
    hipMalloc((void**)&d_W,  size_W  * sizeof(float));
    hipMalloc((void**)&d_hb, size_hb * sizeof(float));

	hipMemcpy(d_V,  V,  size_V_batch * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W,  W,  size_W  * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_hb, hb, size_hb * sizeof(float), hipMemcpyHostToDevice);
   
	crbmInfer(d_H, d_HS, d_V, d_W, d_hb, pars);
    
    hipMemcpy(H,  d_H,  size_H_batch * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(HS, d_HS, size_H_batch * sizeof(float), hipMemcpyDeviceToHost);
   
    hipFree(d_H);
    hipFree(d_HS);
    hipFree(d_V);
	hipFree(d_W);
	hipFree(d_hb);
}

void crbmRecon_proxy(float *V, float *H, float *W, float *vb, Parameters pars)
{
    float *d_V, *d_H, *d_W, *d_vb;
	
	size_t size_V_batch = pars.nv * pars.nv * pars.Kin  * pars.batchsize;
	size_t size_H_batch = pars.nh * pars.nh * pars.Kout * pars.batchsize;
    size_t size_W  = pars.nw * pars.nw * pars.Kin * pars.Kout;
    size_t size_vb = pars.Kin;

	hipMalloc((void**)&d_V,  size_V_batch  * sizeof(float));
	hipMalloc((void**)&d_H,  size_H_batch  * sizeof(float));
    hipMalloc((void**)&d_W,  size_W  * sizeof(float));
    hipMalloc((void**)&d_vb, size_vb * sizeof(float));

	hipMemcpy(d_H,  H,  size_H_batch * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W,  W,  size_W  * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vb, vb, size_vb * sizeof(float), hipMemcpyHostToDevice);
   
	crbmRecon(d_V, d_H, d_W, d_vb, pars);
    
    hipMemcpy(V,  d_V,  size_V_batch * sizeof(float), hipMemcpyDeviceToHost);
   
    hipFree(d_V);
    hipFree(d_H);
	hipFree(d_W);
	hipFree(d_vb);
}

void crbmUpdates_proxy(float *dW, float *dvb, float *dhb, float *V, float *W, float *vb, float *hb, Parameters pars)
{
    float *d_V, *d_dW, *d_dvb, *d_dhb, *d_W, *d_vb, *d_hb;
	
	size_t size_V_batch = pars.nv * pars.nv * pars.Kin  * pars.batchsize;
    size_t size_W  = pars.nw * pars.nw * pars.Kin * pars.Kout;
    size_t size_vb = pars.Kin;
	size_t size_hb = pars.Kout;

	hipMalloc((void**)&d_V,  size_V_batch  * sizeof(float));
    hipMalloc((void**)&d_W,  size_W  * sizeof(float));
    hipMalloc((void**)&d_vb, size_vb * sizeof(float));
	hipMalloc((void**)&d_hb, size_hb * sizeof(float));
    hipMalloc((void**)&d_dW,  size_W  * sizeof(float));
    hipMalloc((void**)&d_dvb, size_vb * sizeof(float));
	hipMalloc((void**)&d_dhb, size_hb * sizeof(float));

	hipMemcpy(d_V,  V,  size_V_batch * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W,  W,  size_W  * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vb, vb, size_vb * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_hb, hb, size_hb * sizeof(float), hipMemcpyHostToDevice);   

	crbmUpdates(d_dW, d_dvb, d_dhb, d_V, d_W, d_vb, d_hb, pars);
    
    hipMemcpy(dW,  d_dW,  size_W  * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(dvb, d_dvb, size_vb * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(dhb, d_dhb, size_hb * sizeof(float), hipMemcpyDeviceToHost);
   
    hipFree(d_V);
	hipFree(d_W);
	hipFree(d_vb);
	hipFree(d_hb);
	hipFree(d_dW);
	hipFree(d_dvb);
	hipFree(d_dhb);
}

void computeVisHid_proxy(float *prods, float *V, float *H, Parameters pars)
{
    float *d_prods, *d_V, *d_H;
	
	size_t size_V_batch = pars.nv * pars.nv * pars.Kin  * pars.batchsize;
    size_t size_H_batch = pars.nh * pars.nh * pars.Kout * pars.batchsize;
	size_t size_prods = pars.nw * pars.nw * pars.Kin * pars.Kout;

	hipMalloc((void**)&d_V, size_V_batch * sizeof(float));
    hipMalloc((void**)&d_H, size_H_batch * sizeof(float));
	hipMalloc((void**)&d_prods, size_prods * sizeof(float));

	hipMemcpy(d_V, V, size_V_batch * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_H, H, size_H_batch * sizeof(float), hipMemcpyHostToDevice);   

	computeVisHid(d_prods, d_V, d_H, pars);
 
	hipMemcpy(prods, d_prods, size_prods * sizeof(float), hipMemcpyDeviceToHost);
   
    hipFree(d_prods);
	hipFree(d_V);
	hipFree(d_H);
}

void computeAct_proxy(float *act, float *batches, int imsize, int channels, int batchsize)
{
    float *d_act, *d_batches;
	
	size_t size_batches = imsize * imsize * channels * batchsize;
    size_t size_act = channels;

	hipMalloc((void**)&d_act, size_act * sizeof(float));
    hipMalloc((void**)&d_batches, size_batches * sizeof(float));
	
	hipMemcpy(d_batches, batches, size_batches * sizeof(float), hipMemcpyHostToDevice);

	computeAct(d_act, d_batches, imsize, channels, batchsize);	

	hipMemcpy(act, d_act, size_act * sizeof(float), hipMemcpyDeviceToHost);
   
    hipFree(d_act);
	hipFree(d_batches);
}




